#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#include "dct_quantize.h"
#include "mjpeg_encoder.h"
/* We import precalculated cos values */
#include "cosv.h"

/* Global variables, for less redundancy processing */
uint32_t uv_width;
uint32_t uv_height;
uint32_t y_out_size;
uint32_t uv_out_size;

int16_t *Ydst;
int16_t *Udst;
int16_t *Vdst;

hipArray *Y;
hipArray *U;
hipArray *V;

/* Init blocks and threads for GPU,
 * One grid for Y, and one for U and V
 */
dim3 block_grid_Y, block_grid_UV, thread_grid;

/* Prepare a variable for the block in texture memory */
texture<uint8_t, 2, hipReadModeElementType> ImgSrc;

/* Just stacking the quantixation tables together.
 * We can use offsets when accessing. This table will not
 * be changes, so we put it in constant memory
 */
__constant__ float quanttbl_gpu[192] =
{
  16, 11, 12, 14, 12, 10, 16, 14,
  13, 14, 18, 17, 16, 19, 24, 40,
  26, 24, 22, 22, 24, 49, 35, 37,
  29, 40, 58, 51, 61, 30, 57, 51,
  56, 55, 64, 72, 92, 78, 64, 68,
  87, 69, 55, 56, 80, 109, 81, 87,
  95, 98, 103, 104, 103, 62, 77, 113,
  121, 112, 100, 120, 92, 101, 103, 99, /*___*/
  17, 18, 18, 24, 21, 24, 47, 26,
  26, 47, 99, 66, 56, 66, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99, /*___*/
  17, 18, 18, 24, 21, 24, 47, 26,
  26, 47, 99, 66, 56, 66, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99,
  99, 99, 99, 99, 99, 99, 99, 99
};

/* The temporary block should go in shared memory... Much faster than global! :) */
__shared__ float tmp_block[64];

/* The DCT is performed on the device. Use the DCT algorithm from precode.
 * ONE thread will work on ONE pixel.
 */
__global__ void dct_quantize(int16_t *out_data, uint32_t padwidth, uint32_t width, uint32_t quant_offset)
{
  int i,j;
  float dct = 0;
  int yb = by * 8;
  int xb = bx * 8;

  /* Get the appropriate quantization table, by offset into quanttbl_gpu. */
  float *quant = &quanttbl_gpu[quant_offset << 6];

  /* Get pixel from texture memory and put it in shared memory */
  tmp_block[(ty << 3) + tx] = tex2D(ImgSrc, (xb+tx), (yb+ty));
  /* Sync all threads, and kick off the 8x8 blocks */
  __syncthreads();

  for(i = 0; i < BLOCK_SIZE; i++)
  {
    for(j = 0; j < BLOCK_SIZE; j++)
      dct += (tmp_block[(i<<3)+j]-128.0f)*COSUV(tx,ty,i,j);
  }

  float a1 = !(ty) ? M_SQRT1_2 : 1.0f;
  float a2 = !(tx) ? M_SQRT1_2 : 1.0f;

  dct *= a1*a2/4.0f;

  out_data[(yb+tx)*padwidth+(xb+ty)] = (int16_t)(floor(0.5f + dct / (quant[tx*BLOCK_SIZE+ty])));
}

/* Init function is called from main in mjpeg_encoder.c. When doing this, we can reduce
 * some overhead, and not have to start ut and initialize the GPU everytime we call dct.
 *
 * Inti sets up grids and thread grids for the YUV format.
 * We also set up necessary variables for processing the frames
 * Init is done by the host!
 */
__host__ void init()
{
  /* Block grid: NUM_8x8BLOCKSxNUM_8x8BLOCKS Y component */
  block_grid_Y.y    = height >> BLOCK_SIZE_LOG;
  block_grid_Y.x    = width >> BLOCK_SIZE_LOG;

  /* Block grid: NUM_8x8BLOCKSxNUM_8x8BLOCKS U and V component */
  block_grid_UV.y   = uph >> BLOCK_SIZE_LOG;
  block_grid_UV.x   = upw >> BLOCK_SIZE_LOG;

  /* Grid size: 8x8 pixels */
  thread_grid.x	    = BLOCK_SIZE;
  thread_grid.y	    = BLOCK_SIZE;

  uv_width	    = (width / 2);
  uv_height	    = (height / 2);

  /* Multiply by 2, since ipupt is uint8_t and output is int16_t */
  uv_out_size	    = (uv_comp_size * 2);
  y_out_size	    = (y_comp_size  * 2);


  /* Handle u and v components, not the same size as Y */
  int uv_width_pad  = (ypw / 2);
  int uv_height_pad = (yph / 2);
  int uv_areal_pad  = (upw * uph);
  int uv_areal_pad2 = (uv_areal_pad * 2);

  /* Do the memory! */

  /* The texture memory for the source image */
  hipChannelFormatDesc chartex = hipCreateChannelDesc<uint8_t>();

  /* Allocate memory on the device for the input data (YUV components) */
  hipMallocArray(&Y, &chartex, width, height);
  hipMallocArray(&U, &chartex, uv_width_pad, uv_height_pad);
  hipMallocArray(&V, &chartex, uv_width_pad, uv_height_pad);

  /* If we must pad, set all values to 128 */
  hipMemset(U,128,uv_areal_pad);
  hipMemset(V,128,uv_areal_pad);

  /* Allocate memory on the device for the output */
  hipMalloc((void **) &Ydst, y_out_size);
  hipMalloc((void **) &Udst, uv_areal_pad2);
  hipMalloc((void **) &Vdst, uv_areal_pad2);
}

/* Host code. This is the function called from mjpeg_encoder.c.
 * Handles all copying, the binding of texture memory and calls
 * the dct_quantize function. It also handles the output data.
 */
__host__ void gpu_dct_quantize(yuv_t *image, dct_t *out)
{
  /* Copy input to the device */
  hipMemcpy2DToArray(Y, 0, 0, image->Y, width * sizeof(uint8_t), width * sizeof(uint8_t), height, hipMemcpyHostToDevice);
  hipMemcpy2DToArray(U, 0, 0, image->U, uv_width * sizeof(uint8_t), uv_width * sizeof(uint8_t), uv_height, hipMemcpyHostToDevice);
  hipMemcpy2DToArray(V, 0, 0, image->V, uv_width * sizeof(uint8_t), uv_width * sizeof(uint8_t), uv_height, hipMemcpyHostToDevice);

  /* Bind array to texture memory and call dct_quantize with Y grid size */
  hipBindTextureToArray(ImgSrc, Y);
  dct_quantize<<<block_grid_Y,  thread_grid>>>(Ydst, ypw, width, Y_QUANT);
  hipUnbindTexture(ImgSrc);

  /* Bind array to texture memory and call dct_quantize with UV (U) grid size */
  hipBindTextureToArray(ImgSrc, U);
  dct_quantize<<<block_grid_UV, thread_grid>>>(Udst, upw, uv_width, U_QUANT);
  hipUnbindTexture(ImgSrc);

  /* Bind array to texture memory and call dct_quantize with UV (V) grid size */
  hipBindTextureToArray(ImgSrc, V);
  dct_quantize<<<block_grid_UV, thread_grid>>>(Vdst, vpw, uv_width, V_QUANT);
  hipUnbindTexture(ImgSrc);

  /* Copy back to the host from the device memory  */
  hipMemcpy(out->Ydct, Ydst, y_out_size,  hipMemcpyDeviceToHost);
  hipMemcpy(out->Udct, Udst, uv_out_size, hipMemcpyDeviceToHost);
  hipMemcpy(out->Vdct, Vdst, uv_out_size, hipMemcpyDeviceToHost);
}

/* In my office, we allways have to clean up! ;) */
__host__ void cleanup()
{
  hipFreeArray(Y);
  hipFreeArray(U);
  hipFreeArray(V);

  hipFree(Ydst);
  hipFree(Udst);
  hipFree(Vdst);
}
